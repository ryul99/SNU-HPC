#include "hip/hip_runtime.h"
#include "matmul.h"
#include "util.h"

#include <hip/hip_runtime.h>
#include <mpi.h>

#define CUDA_CALL(f)                                                           \
  {                                                                            \
    hipError_t err = (f);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at [%s:%d] %d %s\n", __FILE__, __LINE__,     \
              err, hipGetErrorString(err));                                   \
      exit(1);                                                                 \
    }                                                                          \
  }


#define DEBUG 0
#define NUM_ELEM 4096
#define NUM_BUFFER_ELEM 32
#define TS 8
#define BLOCK_ROWS 4
#define NUM_GPU 4
#define NUM_NODE 4
#define NUM_THREAD 256
#define NUM_OUTER_LOOP 4
#define NUM_INNER_LOOP 1

float *h_A[NUM_OUTER_LOOP], *h_B, *h_C;
float *d_A[NUM_OUTER_LOOP][NUM_GPU], *d_B[NUM_GPU], *d_C[NUM_OUTER_LOOP][NUM_GPU];
hipStream_t s_d[NUM_GPU][NUM_INNER_LOOP];
hipEvent_t ev_d[NUM_GPU];
int mpi_rank, mpi_world_size;
MPI_Request req[NUM_OUTER_LOOP];


__global__ void transposeFineGrained(float *dst, const float *src, const int width, const int height)
{
  // ref: https://developer.download.nvidia.com/assets/cuda/files/MatrixTranspose.pdf
  // x: TS, y: BLOCK_ROWS
  __shared__ float block[TS][TS + 1];
  int xIndex = blockIdx.x * TS + threadIdx.x;
  int yIndex = blockIdx.y * TS + threadIdx.y;
  int index = xIndex + (yIndex) * width;

  for (int i=0; i < TS; i += BLOCK_ROWS) {
    block[threadIdx.y+i][threadIdx.x] = src[index+i*width];
  }
  __syncthreads();
  for (int i=0; i < TS; i += BLOCK_ROWS) {
    dst[index+i*height] = block[threadIdx.x][threadIdx.y+i];
  }
}

__global__ void matmul_cal(const float *A, const float *B, float *C, int M, int N, int K) {
  // TODO: FILL_IN_HERE

  // A: M x K
  // B: K x N
  // C: M x N

  // Ap: K x M
  // Bp: N x K

  // 0 ... col_size
  const int col = threadIdx.x;
  // 0 ... row_size
  const int row = threadIdx.y;
  // const int col_size = blockDim.x;
  // const int row_size = blockDim.y;
  // n - col idx
  const int global_col = TS * blockIdx.x + col;
  // m - row idx
  const int global_row = TS * blockIdx.y + row;

  __shared__ float Asub[TS][TS];
  __shared__ float Bsub[TS][TS];

  #if DEBUG
  if (row == 0 && col == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
    for (int row = 0; row < TS; ++row) {
      for (int col = 0; col < TS; ++col) {
        Asub[row][col] = 0;
      }
    }
  }
  #endif

  float c = 0.0;
  const int numTiles = K / TS;
  for (int t = 0; t < numTiles; ++t) {
    const int tiledRow = TS * t + row;
    const int tiledCol = TS * t + col;
    Asub[row][col] = A[tiledCol + K * global_row];
    Bsub[row][col] = B[global_col + N * tiledRow];

    __syncthreads();

    #if DEBUG
    if (row == 0 && col == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
      for (int r = 0; r < TS; ++r) {
        for (int c = 0; c < TS; ++c) {
          if (Asub[r][c] == 0) {
            printf("%d %d\n", r, c);
            // printf("%f\n", Asub[row][col]);
          }
        }
      }
    }
    #endif

    for(int k = 0; k < TS; k++) {
      c += Asub[row][k] * Bsub[k][col];
    }

    __syncthreads();
  }

  C[global_col + N * global_row] = c;
}


void matmul(const float *A, const float *B, float *C, int M, int N, int K) {
  // TODO: FILL_IN_HERE
  // A: M x K
  // B: K x N
  // C: M x N
  //
  // M: Outer -> Node -> GPU -> Inner

  h_B = (float *) B;
  memset(h_C, 0, sizeof(float) * M * N);

  const int nodeM = M / NUM_NODE / NUM_OUTER_LOOP;

  MPI_Bcast(h_B, K * N, MPI_FLOAT, 0, MPI_COMM_WORLD);
 
  for (int d = 0; d < NUM_GPU; ++d) {
    CUDA_CALL(hipSetDevice(d));
    
    CUDA_CALL(hipMemcpyAsync(
      d_B[d], h_B, sizeof(float) * K * N, hipMemcpyHostToDevice, s_d[d][0]
    ));
  }
  for (int d = 0; d < NUM_GPU; ++d) {
    CUDA_CALL(hipSetDevice(d));
    CUDA_CALL(hipStreamSynchronize(s_d[d][0]));
  }

  MPI_Iscatter(
    &A[0], K * nodeM, MPI_FLOAT,
    h_A[0], K * nodeM, MPI_FLOAT,
    0, MPI_COMM_WORLD, &req[0]
  );

  for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
    #if DEBUG
    if (mpi_rank == 0) {
      printf("l: %d\n", l);
    }
    #endif

    if (l < NUM_OUTER_LOOP - 1) {
      MPI_Iscatter(
        &A[K * nodeM * NUM_NODE * (l + 1)], K * nodeM, MPI_FLOAT,
        h_A[(l + 1)], K * nodeM, MPI_FLOAT,
        0, MPI_COMM_WORLD, &req[(l + 1)]
      );
    }
    
    MPI_Wait(&req[l], MPI_STATUSES_IGNORE);

    for (int d = 0; d < NUM_GPU; ++d) {
      CUDA_CALL(hipSetDevice(d));
      
      const int perM = nodeM / NUM_GPU / NUM_INNER_LOOP;

      for (int s = 0; s < NUM_INNER_LOOP; ++s) {
        CUDA_CALL(hipMemcpyAsync(
          &d_A[l][d][(s * perM) * K],
          &h_A[l][(s * perM + d * perM * NUM_INNER_LOOP) * K],
          sizeof(float) * perM * K, hipMemcpyHostToDevice, s_d[d][s]
        ));


        dim3 dimBlock(TS, TS);
        dim3 dimGrid(N / TS, perM / TS);

        #if DEBUG
        if (mpi_rank == 0) {
          printf("dimBlock: %d %d\n", TS, TS);
          printf("dimGrid: %d %d\n", N / TS, perM / TS);
        }
        #endif

        matmul_cal<<<dimGrid, dimBlock, 0, s_d[d][s]>>>(
          &d_A[l][d][(s * perM) * K], d_B[d], &d_C[l][d][(s * perM) * N], perM , N, K
        );
        CUDA_CALL(hipMemcpyAsync(
          &h_C[(s * perM + d * perM * NUM_INNER_LOOP + mpi_rank * nodeM + l * M / NUM_OUTER_LOOP) * N], &d_C[l][d][(s * perM) * N],
          sizeof(float) * perM * N, hipMemcpyDeviceToHost,
          s_d[d][s]
        ));
      }
    }
  }
  
  for (int d = 0; d < NUM_GPU; ++d) {
    CUDA_CALL(hipSetDevice(d));
    for (int i = 0; i < NUM_INNER_LOOP; ++i) {
      CUDA_CALL(hipStreamSynchronize(s_d[d][i]));
    }
    CUDA_CALL(hipDeviceSynchronize());
  }

  MPI_Reduce(h_C, C, M * N, MPI_FLOAT, MPI_SUM, 0, MPI_COMM_WORLD);
}

void matmul_initialize(int M, int N, int K) {
  // TODO: FILL_IN_HERE
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);

  for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
    CUDA_CALL(hipHostMalloc(&h_A[l], sizeof(float) * M * K / NUM_NODE / NUM_OUTER_LOOP));
  }
  CUDA_CALL(hipHostMalloc(&h_B, sizeof(float) * K * N));
  CUDA_CALL(hipHostMalloc(&h_C, sizeof(float) * M * N));

  for (int d = 0; d < NUM_GPU; ++d) {
    CUDA_CALL(hipSetDevice(d));
    CUDA_CALL(hipMalloc(&d_B[d], sizeof(float) * K * N));
    for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
      CUDA_CALL(hipMalloc(&d_A[l][d], sizeof(float) * M * K / NUM_NODE / NUM_GPU / NUM_OUTER_LOOP));
      CUDA_CALL(hipMalloc(&d_C[l][d], sizeof(float) * M * N / NUM_NODE / NUM_GPU / NUM_OUTER_LOOP));  
    }
    for (int l = 0; l < NUM_INNER_LOOP; ++l) {
      CUDA_CALL(hipStreamCreate(&s_d[d][l]));
    }
    CUDA_CALL(hipEventCreate(&ev_d[d]));
  }
}

void matmul_finalize() {
  // TODO: FILL_IN_HERE
  for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
    CUDA_CALL(hipHostFree(h_A[l]));
  }
  CUDA_CALL(hipHostFree(h_B));
  CUDA_CALL(hipHostFree(h_C));

  for (int d = 0; d < NUM_GPU; ++d) {
    CUDA_CALL(hipSetDevice(d));
    CUDA_CALL(hipFree(d_B[d]));
    for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
      CUDA_CALL(hipFree(d_A[l][d]));
      CUDA_CALL(hipFree(d_C[l][d]));
    }
    for (int l = 0; l < NUM_INNER_LOOP; ++l) {
      CUDA_CALL(hipStreamDestroy(s_d[d][l]));
    }
    CUDA_CALL(hipEventDestroy(ev_d[d]));
  }
}
