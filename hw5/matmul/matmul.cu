#include "hip/hip_runtime.h"
#include "matmul.h"
#include "util.h"

#include <hip/hip_runtime.h>
#include <mpi.h>

#define CUDA_CALL(f)                                                           \
  {                                                                            \
    hipError_t err = (f);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at [%s:%d] %d %s\n", __FILE__, __LINE__,     \
              err, hipGetErrorString(err));                                   \
      exit(1);                                                                 \
    }                                                                          \
  }


#define MIN(a, b) ((a) < (b) ? (a) : (b))


#define DEBUG 0
#define KERNEL_DEBUG 0
#define SUMMARY 0
#define TS 32
#define BLOCK_ROWS 4
#define NUM_GPU 4
#define NUM_NODE 4
#define NUM_OUTER_LOOP 16
#define DIV_STREAM 1024
#define NUM_FUSION 1
#define NUM_MPI 2
// NUM_MPI should be devidor of NUM_OUTER_LOOP
#if NUM_MPI >= 1
  #define MPI_TS (NUM_OUTER_LOOP / NUM_MPI)
#else
  #define MPI_TS 1
#endif

#if NUM_MPI >= 1
float *h_A_buff[NUM_MPI];
#endif

float *h_A[NUM_OUTER_LOOP], *h_B, *h_C;
float *d_A[NUM_OUTER_LOOP / NUM_FUSION][NUM_GPU], *d_B[NUM_GPU], *d_C[NUM_OUTER_LOOP / NUM_FUSION][NUM_GPU];
hipStream_t s_d[NUM_GPU][NUM_OUTER_LOOP / NUM_FUSION][3];
hipEvent_t ev_buff[NUM_GPU][NUM_OUTER_LOOP / NUM_FUSION][2];
int mpi_rank, mpi_world_size;
MPI_Request req[NUM_OUTER_LOOP], reqB;


struct matmul_args {
  int M;
  int N;
  int K;
  float *C;
};



__global__ void transposeFineGrained(float *dst, const float *src, const int width, const int height)
{
  // ref: https://developer.download.nvidia.com/assets/cuda/files/MatrixTranspose.pdf
  // x: TS, y: BLOCK_ROWS
  __shared__ float block[TS][TS + 1];
  int xIndex = blockIdx.x * TS + threadIdx.x;
  int yIndex = blockIdx.y * TS + threadIdx.y;
  int index = xIndex + (yIndex) * width;

  for (int i=0; i < TS; i += BLOCK_ROWS) {
    block[threadIdx.y+i][threadIdx.x] = src[index+i*width];
  }
  __syncthreads();
  for (int i=0; i < TS; i += BLOCK_ROWS) {
    dst[index+i*height] = block[threadIdx.x][threadIdx.y+i];
  }
}

__global__ void matmul_cal(const float *A, const float *B, float *C, int M, int N, int K) {
  // TODO: FILL_IN_HERE

  // A: M x K
  // B: K x N
  // C: M x N

  // Ap: K x M
  // Bp: N x K

  // 0 ... col_size
  const int col = threadIdx.x;
  // 0 ... row_size
  const int row = threadIdx.y;
  // const int col_size = blockDim.x;
  // const int row_size = blockDim.y;
  // n - col idx
  const int global_col = TS * blockIdx.x + col;
  // m - row idx
  const int global_row = TS * blockIdx.y + row;

  // print global_col global_row
  // #if KERNEL_DEBUG
  // if (global_row > 2048)
  //   printf("global_col: %d, global_row: %d\n", global_col, global_row);
  // #endif

  __shared__ float Asub[TS][TS];
  __shared__ float Bsub[TS][TS];

  #if KERNEL_DEBUG
  if (row == 0 && col == 0 && global_row == 0 && global_col == 0) {
    for (int row = 0; row < TS; ++row) {
      for (int col = 0; col < TS; ++col) {
        Asub[row][col] = 0;
        Bsub[row][col] = 0;
      }
    }
    // check bound
    if (global_row >= M || global_col >= N) {
      printf("global_row: %d, global_col: %d\n", global_row, global_col);
    }
  }
  #endif

  float c = 0.0;
  const int numTiles = K / TS;
  for (int t = 0; t < numTiles; ++t) {
    const int tiledRow = TS * t + row;
    const int tiledCol = TS * t + col;
    Asub[row][col] = A[tiledCol + K * global_row];
    Bsub[row][col] = B[global_col + N * tiledRow];

    __syncthreads();

    #if KERNEL_DEBUG
    if (Asub[row][col] == 0) {
      printf("Asub[%d][%d] = %f and Grid Idx: [%d][%d]\n", row, col, Asub[row][col], blockIdx.x * TS, blockIdx.y * TS);
    }
    if (Bsub[row][col] == 0) {
      printf("Bsub[%d][%d] = %f and Grid Idx: [%d][%d]\n", row, col, Bsub[row][col], blockIdx.x * TS, blockIdx.y * TS);
    }
    #endif

    for(int k = 0; k < TS; k++) {
      c += Asub[row][k] * Bsub[k][col];
    }

    __syncthreads();
  }
  #if KERNEL_DEBUG
  if (c == 0) {
    printf("C[%d][%d] = %f\n", global_row, global_col, c);
  }
  #endif
  C[global_col + N * global_row] = c;
}


void createEvent() {
  for (int d = 0; d < NUM_GPU; ++d) {
    CUDA_CALL(hipSetDevice(d));
    for (int l = 0; l < NUM_OUTER_LOOP / NUM_FUSION; ++l) {
      for (int i = 0; i < 2; ++i)
        CUDA_CALL(hipEventCreate(&ev_buff[d][l][i]));
    }
  }
}

void createStream() {
  for (int d = 0; d < NUM_GPU; ++d) {
    CUDA_CALL(hipSetDevice(d));
    for (int l = 0; l < NUM_OUTER_LOOP / NUM_FUSION; ++l) {
      for (int i = 0; i < 3; ++i)
        CUDA_CALL(hipStreamCreate(&s_d[d][l][i]));
    }
  }
}

void destroyEvent() {
  for (int d = 0; d < NUM_GPU; ++d) {
    CUDA_CALL(hipSetDevice(d));
    for (int l = 0; l < NUM_OUTER_LOOP / NUM_FUSION; ++l) {
      for (int i = 0; i < 2; ++i)
        CUDA_CALL(hipEventDestroy(ev_buff[d][l][i]));
    }
  }
}

void destroyStream() {
  for (int d = 0; d < NUM_GPU; ++d) {
    CUDA_CALL(hipSetDevice(d));
    for (int l = 0; l < NUM_OUTER_LOOP / NUM_FUSION; ++l) {
      for (int i = 0; i < 3; ++i)
        CUDA_CALL(hipStreamDestroy(s_d[d][l][i]));
    }
  }
}


void loadA(int K, int perM, int d, int l) {
  CUDA_CALL(hipSetDevice(d));
  int d_A_d_idx = ((d + l * NUM_GPU) / NUM_FUSION) % NUM_GPU;
  #if DEBUG
  printf("d_A index: %d, in array index: %d, %d\n", l / NUM_FUSION, d_A_d_idx, ((d % NUM_FUSION)));
  printf("h_A index: %d, in array index: %d\n", l, d);
  #endif
  CUDA_CALL(hipMemcpyAsync(
    &d_A[l / NUM_FUSION][d_A_d_idx][(d % NUM_FUSION) * perM * K],
    &h_A[l][d * perM * K],
    sizeof(float) * perM * K, hipMemcpyHostToDevice, s_d[d_A_d_idx][l / NUM_FUSION][0]
  ));
  #if DEBUG
  printf("ev_buff[%d][%d][0]\n", d_A_d_idx, l / NUM_FUSION);
  printf("s_d[%d][%d][0]\n", d_A_d_idx, l / NUM_FUSION);
  printf("\n\n");
  #endif
  CUDA_CALL(hipEventRecord(ev_buff[d_A_d_idx][l / NUM_FUSION][0], s_d[d_A_d_idx][l / NUM_FUSION][0]));
}


void matmul(const float *A, const float *B, float *C, int M, int N, int K) {
  // TODO: FILL_IN_HERE
  // A: M x K
  // B: K x N
  // C: M x N
  //
  // M: Outer -> Node -> GPU

  const int nodeM = M / NUM_NODE / NUM_OUTER_LOOP;
  const int perM = nodeM / NUM_GPU;
  #if NUM_MPI > 1
  int perNodeM = M / NUM_NODE / NUM_MPI;
  #endif

  #if SUMMARY
  if (mpi_rank == 0) {
    const int check = NUM_NODE * NUM_OUTER_LOOP * NUM_GPU * NUM_FUSION;
    printf("dimBlock: %d %d\n", TS, TS);
    printf("dimGrid: %d %d\n", N / TS, NUM_FUSION * perM / TS);
    printf("NUM_FUSION: %d\n", NUM_FUSION);
    printf("perM: %d\n", perM);
    printf("NUM_OUTER_LOOP: %d\n", NUM_OUTER_LOOP);
    printf("M should be bigger than %d and... validation: %d\n", check, M / check);
  }
  #endif

  createEvent();
  createStream();

  h_B = (float *) B;
  // memset(h_C, 0, sizeof(float) * M * N);

  #if NUM_MPI > 0
  MPI_Ibcast(h_B, K * N, MPI_FLOAT, 0, MPI_COMM_WORLD, &reqB);
  #if NUM_MPI == 1
  for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
    h_A[l] = (float *) &h_A_buff[K * nodeM * l];
  }
  MPI_Iscatter(
    A, M * K / NUM_NODE, MPI_FLOAT,
    h_A_buff[0], M * K / NUM_NODE, MPI_FLOAT,
    0, MPI_COMM_WORLD, &req[0]
  );
  #else
  for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
    h_A[l] = (float *) &h_A_buff[l / MPI_TS][K * nodeM * (l % MPI_TS)];
  }
  for (int l = 0; l < NUM_MPI; ++l) {
    MPI_Iscatter(
      &A[K * perNodeM * NUM_NODE * l], K * perNodeM, MPI_FLOAT,
      h_A_buff[l], K * perNodeM, MPI_FLOAT,
      0, MPI_COMM_WORLD, &req[l]
    );
  }
  #endif
  #else
  for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
    h_A[l] = (float *) &A[K * nodeM * l];
  }
  h_C = C;
  #endif
  #if NUM_MPI > 0
  MPI_Wait(&reqB, MPI_STATUS_IGNORE);
  #if NUM_MPI == 1
  MPI_Wait(&req[0], MPI_STATUS_IGNORE);
  #endif
  #endif
  for (int d = 0; d < NUM_GPU; ++d) {
    CUDA_CALL(hipSetDevice(d));
    
    CUDA_CALL(hipMemcpyAsync(
      d_B[d], h_B, sizeof(float) * K * N, hipMemcpyHostToDevice, s_d[d][0][0]
    ));
  }
  for (int d = 0; d < NUM_GPU; ++d) {
    CUDA_CALL(hipSetDevice(d));
    CUDA_CALL(hipStreamSynchronize(s_d[d][0][0]));
  }

  #if NUM_MPI > 0
  #pragma omp parallel for
  #endif
  for (int t = 0; t < NUM_OUTER_LOOP / NUM_FUSION; t += MPI_TS) {
    #if NUM_MPI
    #if (MPI_TS) >= NUM_FUSION
    MPI_Wait(
      &req[t / ((MPI_TS) / NUM_FUSION)], MPI_STATUS_IGNORE
    );
    #endif
    #endif
    for (int l = t; l < MIN(t + MPI_TS, NUM_OUTER_LOOP / NUM_FUSION); ++l) {
      #if NUM_MPI > 0
      #if (MPI_TS) < NUM_FUSION
      MPI_Waitall(
        NUM_FUSION / (MPI_TS),
        &req[l * NUM_FUSION / (MPI_TS)], MPI_STATUSES_IGNORE
      );
      #endif
      #endif

      for (int d = 0; d < NUM_GPU; ++d) {
        CUDA_CALL(hipSetDevice(d));
        for (int i = 0; i < NUM_FUSION; ++i) {
          loadA(K, perM, d, l * NUM_FUSION + i);
        }
        #if DEBUG
        printf("s_d_Cal index: %d\n", l % DIV_STREAM);
        #endif
        dim3 dimBlock(TS, TS);
        dim3 dimGrid(N / TS, NUM_FUSION * perM / TS);
        CUDA_CALL(hipStreamWaitEvent(s_d[d][l % DIV_STREAM][1], ev_buff[d][l][0]));

        matmul_cal<<<dimGrid, dimBlock, 0, s_d[d][l % DIV_STREAM][1]>>>(
          d_A[l][d], d_B[d], d_C[l][d], perM * NUM_FUSION, N, K
        );
        CUDA_CALL(hipGetLastError());

        #if DEBUG
        printf("d_C index: %d, %d\n", l, d);
        #endif

        CUDA_CALL(hipEventRecord(ev_buff[d][l][1], s_d[d][l % DIV_STREAM][1]));

        CUDA_CALL(hipStreamWaitEvent(s_d[d][l % DIV_STREAM][2], ev_buff[d][l][1]));
        CUDA_CALL(hipMemcpyAsync(
          &h_C[((d * perM + (l) * nodeM) * NUM_FUSION) * N], d_C[l][d],
          sizeof(float) * perM * N * NUM_FUSION, hipMemcpyDeviceToHost,
          s_d[d][l % DIV_STREAM][2]
        ));
        #if DEBUG
        printf("h_C index: %d\n", ((d * perM + (l) * nodeM) * NUM_FUSION));
        #endif

        #if DEBUG
        printf("\n\n");
        #endif
      }
    }
  }
  #if NUM_MPI > 0
  #if NUM_MPI == 1
  MPI_Gather(
    h_C, M * N / NUM_NODE, MPI_FLOAT,
    C, M * N / NUM_NODE, MPI_FLOAT,
    0, MPI_COMM_WORLD
  );
  #else
  #pragma omp parallel for
  for (int l = 0; l < NUM_MPI; ++l) {
    for (int ll = l * MPI_TS; ll < MIN(NUM_OUTER_LOOP, (l + 1) * MPI_TS); ++ll) {
      int lll = ll / NUM_FUSION;
      for (int d = 0; d < NUM_GPU; ++d) {
        CUDA_CALL(hipSetDevice(d));
        CUDA_CALL(hipStreamSynchronize(s_d[d][lll % DIV_STREAM][2]));
      }
    }
    if (mpi_rank == 0) {
      for (int r = 1; r < mpi_world_size; ++r) {
        MPI_Recv(
          &C[(l * NUM_NODE + r) * perNodeM *  N], perNodeM * N, MPI_FLOAT,
          r, r + l * mpi_world_size, MPI_COMM_WORLD, MPI_STATUS_IGNORE
        );
      }
      memcpy(&C[l * NUM_NODE * perNodeM * N], &h_C[l * perNodeM * N], perNodeM * N * sizeof(float));
    } else {
      MPI_Send(
        &h_C[l * perNodeM * N], perNodeM * N, MPI_FLOAT,
        0, mpi_rank + l * mpi_world_size, MPI_COMM_WORLD
      );
    } 
  }
  #endif
  #else
  for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
    for (int d = 0; d < NUM_GPU; ++d) {
      CUDA_CALL(hipSetDevice(d));
      CUDA_CALL(hipStreamSynchronize(s_d[d][l / NUM_FUSION % DIV_STREAM][2]));
    }
  }
  #endif
  destroyEvent();
  destroyStream();
}

void warn_values() {
  // check NUM_NODE, NUM_GPU, NUM_FUSION, NUM_OUTER_LOOP with each rank
  if (NUM_NODE != mpi_world_size)
    printf("[WARN] (rank: %d) NODE => set: %d / current active: %d\n", mpi_rank, NUM_NODE, mpi_world_size);
  int num_gpu;
  CUDA_CALL(hipGetDeviceCount(&num_gpu));
  if (NUM_GPU != num_gpu)
    printf("[WARN] (rank: %d) GPU => set: %d / current active: %d\n", mpi_rank, NUM_GPU, num_gpu);
  if (NUM_OUTER_LOOP % NUM_FUSION != 0 | NUM_OUTER_LOOP < NUM_FUSION)
    printf("[WARN] (rank: %d) NUM_OUTER_LOOP: %d, NUM_FUSION: %d\n", mpi_rank, NUM_OUTER_LOOP, NUM_FUSION);
  #if NUM_MPI
  if (NUM_OUTER_LOOP % (NUM_MPI * NUM_NODE) != 0 | NUM_OUTER_LOOP < NUM_MPI * NUM_NODE)
    printf("[WARN] (rank: %d) NUM_OUTER_LOOP: %d, NUM_MPI: %d, NUM_NODE: %d\n", mpi_rank, NUM_OUTER_LOOP, NUM_MPI, NUM_NODE);
  if (! (NUM_FUSION % MPI_TS == 0 || MPI_TS % NUM_FUSION == 0))
    printf("[WARN] (rank: %d) NUM_FUSION: %d, MPI_TS: %d\n", mpi_rank, NUM_FUSION, MPI_TS);
  #endif
}

void matmul_initialize(int M, int N, int K) {
  // TODO: FILL_IN_HERE
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);

  warn_values();

  // print NUM GPU
  #if SUMMARY
  int num_gpu;
  CUDA_CALL(hipGetDeviceCount(&num_gpu));
  printf("(rank: %d) NUM_GPU: %d\n", mpi_rank, num_gpu);
  #endif
  #if NUM_MPI > 0
  CUDA_CALL(hipHostMalloc(&h_C, sizeof(float) * M * N / NUM_NODE));
  for (int i = 0; i < NUM_MPI; ++i)
    CUDA_CALL(hipHostMalloc(&h_A_buff[i], sizeof(float) * M * K / NUM_NODE / NUM_MPI));
  #endif

  for (int d = 0; d < NUM_GPU; ++d) {
    CUDA_CALL(hipSetDevice(d));
    CUDA_CALL(hipMalloc(&d_B[d], sizeof(float) * K * N));
    for (int l = 0; l < NUM_OUTER_LOOP / NUM_FUSION; ++l) {
      CUDA_CALL(hipMalloc(&d_A[l][d], sizeof(float) * M * K  * NUM_FUSION / NUM_NODE / NUM_GPU / NUM_OUTER_LOOP));
      CUDA_CALL(hipMalloc(&d_C[l][d], sizeof(float) * M * N  * NUM_FUSION / NUM_NODE / NUM_GPU / NUM_OUTER_LOOP));
    }
  }
}

void matmul_finalize() {
  // TODO: FILL_IN_HERE
  #if NUM_MPI > 0
  CUDA_CALL(hipHostFree(h_C));
  for (int i = 0; i < NUM_MPI; ++i)
    CUDA_CALL(hipHostFree(h_A_buff[i]));
  #endif

  for (int d = 0; d < NUM_GPU; ++d) {
    CUDA_CALL(hipSetDevice(d));
    CUDA_CALL(hipFree(d_B[d]));
    for (int l = 0; l < NUM_OUTER_LOOP / NUM_FUSION; ++l) {
      CUDA_CALL(hipFree(d_A[l][d]));
      CUDA_CALL(hipFree(d_C[l][d]));
    }
  }
}
