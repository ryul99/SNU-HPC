#include "hip/hip_runtime.h"
#include "matmul.h"
#include "util.h"

#include <hip/hip_runtime.h>
#include <mpi.h>

#define CUDA_CALL(f)                                                           \
  {                                                                            \
    hipError_t err = (f);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at [%s:%d] %d %s\n", __FILE__, __LINE__,     \
              err, hipGetErrorString(err));                                   \
      exit(1);                                                                 \
    }                                                                          \
  }


#define DEBUG 0
#define SUMMARY 1
#define NUM_ELEM 4096
#define NUM_BUFFER_ELEM 32
#define TS 32
#define BLOCK_ROWS 4
#define NUM_GPU 4
#define NUM_NODE 4
#define NUM_THREAD 256
#define NUM_OUTER_LOOP 8

float *h_A[NUM_OUTER_LOOP], *h_B, *h_C;
float *d_A[NUM_OUTER_LOOP], *d_B, *d_C[NUM_OUTER_LOOP];
hipStream_t s_d[NUM_OUTER_LOOP];
// hipEvent_t ev_buff[NUM_GPU][NUM_INNER_LOOP][2];
hipEvent_t ev_d[NUM_OUTER_LOOP];
int mpi_rank, mpi_world_size, device_id;
MPI_Request req[NUM_OUTER_LOOP];

struct matmul_args {
  int M;
  int N;
  int K;
  float *C;
};



__global__ void transposeFineGrained(float *dst, const float *src, const int width, const int height)
{
  // ref: https://developer.download.nvidia.com/assets/cuda/files/MatrixTranspose.pdf
  // x: TS, y: BLOCK_ROWS
  __shared__ float block[TS][TS + 1];
  int xIndex = blockIdx.x * TS + threadIdx.x;
  int yIndex = blockIdx.y * TS + threadIdx.y;
  int index = xIndex + (yIndex) * width;

  for (int i=0; i < TS; i += BLOCK_ROWS) {
    block[threadIdx.y+i][threadIdx.x] = src[index+i*width];
  }
  __syncthreads();
  for (int i=0; i < TS; i += BLOCK_ROWS) {
    dst[index+i*height] = block[threadIdx.x][threadIdx.y+i];
  }
}

__global__ void matmul_cal(const float *A, const float *B, float *C, int M, int N, int K) {
  // TODO: FILL_IN_HERE

  // A: M x K
  // B: K x N
  // C: M x N

  // Ap: K x M
  // Bp: N x K

  // 0 ... col_size
  const int col = threadIdx.x;
  // 0 ... row_size
  const int row = threadIdx.y;
  // const int col_size = blockDim.x;
  // const int row_size = blockDim.y;
  // n - col idx
  const int global_col = TS * blockIdx.x + col;
  // m - row idx
  const int global_row = TS * blockIdx.y + row;

  __shared__ float Asub[TS][TS];
  __shared__ float Bsub[TS][TS];

  #if DEBUG
  if (row == 0 && col == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
    for (int row = 0; row < TS; ++row) {
      for (int col = 0; col < TS; ++col) {
        Asub[row][col] = 0;
        Bsub[row][col] = 0;
      }
    }
  }
  #endif

  float c = 0.0;
  const int numTiles = K / TS;
  for (int t = 0; t < numTiles; ++t) {
    const int tiledRow = TS * t + row;
    const int tiledCol = TS * t + col;
    Asub[row][col] = A[tiledCol + K * global_row];
    Bsub[row][col] = B[global_col + N * tiledRow];
    
    __syncthreads();

    for(int k = 0; k < TS; k++) {
      c += Asub[row][k] * Bsub[k][col];
    }

    __syncthreads();
  }

  C[global_col + N * global_row] = c;
  #if DEBUG
  printf("%d %d %f\n", global_row, global_col, c);
  #endif
}


void* gather_func(void *args) {
  struct matmul_args *arg = (struct matmul_args *) args;
  const int M = arg->M;
  const int N = arg->N;
  const int K = arg->K;
  float *C = arg->C;

  const int perM = M / NUM_GPU / NUM_NODE / NUM_OUTER_LOOP;

  for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
    // spinlock
    while (hipEventQuery(ev_d[l]) != hipSuccess);
    MPI_Gather(
      &h_C[l * perM * N], perM * N, MPI_FLOAT,
      &C[l * perM * NUM_GPU * NUM_NODE * N], perM * N, MPI_FLOAT,
      0, MPI_COMM_WORLD
    );
  }
  return NULL;
}


void matmul(const float *A, const float *B, float *C, int M, int N, int K) {
  // TODO: FILL_IN_HERE
  // A: M x K
  // B: K x N
  // C: M x N
  //
  // M: Outer -> Node -> GPU

  #if SUMMARY
  if (mpi_rank == 0) {
    const int perM = 4 * NUM_ELEM / NUM_NODE / NUM_OUTER_LOOP / NUM_GPU;
    printf("dimBlock: %d %d\n", TS, TS);
    printf("dimGrid: %d %d\n", NUM_ELEM / TS, perM / TS);
    printf("perM: %d\n", perM);
    printf("NUM_OUTER_LOOP: %d\n", NUM_OUTER_LOOP);
  }
  #endif

  CUDA_CALL(hipSetDevice(device_id));
  
  // create event & stream
  for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
      CUDA_CALL(hipEventCreate(&ev_d[l]));
  }
  for (int st = 0; st < NUM_OUTER_LOOP; ++st) {
    CUDA_CALL(hipStreamCreate(&s_d[st]));
  }

  h_B = (float *) B;
  // memset(h_C, 0, sizeof(float) * M * N);


  const int nodeM = M / NUM_NODE / NUM_OUTER_LOOP;
  const int perM = nodeM / NUM_GPU;

  MPI_Bcast(h_B, K * N, MPI_FLOAT, 0, MPI_COMM_WORLD);

  for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
    MPI_Iscatter(
      &A[K * nodeM * NUM_NODE * l], K * perM, MPI_FLOAT,
      h_A[l], K * perM, MPI_FLOAT,
      0, MPI_COMM_WORLD, &req[l]
    );
  }
  
  CUDA_CALL(hipMemcpyAsync(
    d_B, h_B, sizeof(float) * K * N, hipMemcpyHostToDevice, s_d[0]
  ));
  CUDA_CALL(hipStreamSynchronize(s_d[0]));

  for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
    #if DEBUG
    if (mpi_rank == 0) {
      printf("l: %d\n", l);
    }
    #endif
    
    MPI_Wait(&req[l], MPI_STATUSES_IGNORE);

    CUDA_CALL(hipMemcpyAsync(
      d_A[l],
      h_A[l],
      sizeof(float) * perM * K, hipMemcpyHostToDevice, s_d[l]
    ));


    dim3 dimBlock(TS, TS);
    dim3 dimGrid(N / TS, perM / TS);
    matmul_cal<<<dimGrid, dimBlock, 0, s_d[l]>>>(
      d_A[l], d_B, d_C[l], perM , N, K
    );
    CUDA_CALL(hipGetLastError());

    CUDA_CALL(hipMemcpyAsync(
      &h_C[l * perM * N], d_C[l],
      sizeof(float) * perM * N, hipMemcpyDeviceToHost,
      s_d[l]
    ));

    CUDA_CALL(hipEventRecord(ev_d[l], s_d[l]));
  }
  pthread_t gather_thread;
  struct matmul_args *args = (struct matmul_args *) malloc(sizeof(struct matmul_args));
  args->M = M;
  args->N = N;
  args->K = K;
  args->C = C;
  pthread_create(&gather_thread, NULL, gather_func, args);
  pthread_join(gather_thread, NULL);
  
  // destroy event
  for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
      CUDA_CALL(hipEventDestroy(ev_d[l]));
  }
  // destroy stream
    for (int st = 0; st < NUM_OUTER_LOOP; ++st) {
      CUDA_CALL(hipStreamDestroy(s_d[st]));
    }
  
  #if DEBUG
  if (mpi_rank == 0) {
    printf("dimBlock: %d %d\n", TS, TS);
    printf("dimGrid: %d %d\n", N / TS, perM / TS);
    printf("perM: %d\n", perM);
  }
  #endif
}

void matmul_initialize(int M, int N, int K) {
  // TODO: FILL_IN_HERE
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);
  device_id = mpi_rank % NUM_GPU;
  CUDA_CALL(hipSetDevice(device_id));

  for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
    CUDA_CALL(hipHostMalloc(&h_A[l], sizeof(float) * M * K / NUM_GPU / NUM_NODE / NUM_OUTER_LOOP));
  }
  CUDA_CALL(hipHostMalloc(&h_B, sizeof(float) * K * N));
  CUDA_CALL(hipHostMalloc(&h_C, sizeof(float) * M * N / NUM_GPU / NUM_NODE));

  CUDA_CALL(hipMalloc(&d_B, sizeof(float) * K * N));
  for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
    CUDA_CALL(hipMalloc(&d_A[l], sizeof(float) * M * K / NUM_NODE / NUM_GPU / NUM_OUTER_LOOP));
    CUDA_CALL(hipMalloc(&d_C[l], sizeof(float) * M * N / NUM_NODE / NUM_GPU / NUM_OUTER_LOOP));
  }
}

void matmul_finalize() {
  // TODO: FILL_IN_HERE

  CUDA_CALL(hipSetDevice(device_id));

  #if SUMMARY
  if (mpi_rank == 0) {
    const int perM = 4 * NUM_ELEM / NUM_NODE / NUM_OUTER_LOOP / NUM_GPU;
    printf("dimBlock: %d %d\n", TS, TS);
    printf("dimGrid: %d %d\n", NUM_ELEM / TS, perM / TS);
    printf("perM: %d\n", perM);
  }
  #endif

  for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
    CUDA_CALL(hipHostFree(h_A[l]));
  }
  CUDA_CALL(hipHostFree(h_B));
  CUDA_CALL(hipHostFree(h_C));

  CUDA_CALL(hipFree(d_B));
  for (int l = 0; l < NUM_OUTER_LOOP; ++l) {
    CUDA_CALL(hipFree(d_A[l]));
    CUDA_CALL(hipFree(d_C[l]));
  }
}
