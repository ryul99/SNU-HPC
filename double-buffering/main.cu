#include "hip/hip_runtime.h"
#include <stdio.h>

#define CUDA_CALL(f)                                                           \
  {                                                                            \
    hipError_t err = (f);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at [%s:%d] %d %s\n", __FILE__, __LINE__,     \
              err, hipGetErrorString(err));                                   \
      exit(1);                                                                 \
    }                                                                          \
  }


#define NUM_ELEM (1ul << 28)
#define NUM_BUFFER_ELEM (1ul << 20)

#define EPS (1e-6)

static float *h_A, *h_B, *h_C_naive, *h_C_buffered;
static float *d_A, *d_B, *d_C;
static hipStream_t s0, s1, s2;
static hipEvent_t ev0, ev1;

__global__ void fakeVecAdd(float *A, float *B, float *C, int N)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= N) return;

  C[i] = 0;
  for (int j = 0; j < 128; ++j) {
    C[i] += A[i] + B[i];
  }
}


void fakeVecAdd_naive()
{
  CUDA_CALL(hipMemcpyAsync(
    d_A, h_A, sizeof(float) * NUM_ELEM, hipMemcpyHostToDevice, s0));
  CUDA_CALL(hipMemcpyAsync(
    d_B, h_B, sizeof(float) * NUM_ELEM, hipMemcpyHostToDevice, s0));

  size_t numThreads = 256;
  size_t numBlocks = (NUM_ELEM + numThreads - 1) / numThreads;
  fakeVecAdd<<<numBlocks, numThreads, 0, s0>>>(d_A, d_B, d_C, NUM_ELEM);
  CUDA_CALL(hipGetLastError());

  CUDA_CALL(hipMemcpyAsync(
    h_C_naive, d_C, sizeof(float) * NUM_ELEM, hipMemcpyDeviceToHost, s0));

  CUDA_CALL(hipStreamSynchronize(s0));
}

void fakeVecAdd_double_buffered()
{
  for (int off = 0; off < NUM_ELEM; off += NUM_BUFFER_ELEM) {
    CUDA_CALL(hipMemcpyAsync(
        &d_A[off], &h_A[off], sizeof(float) * NUM_BUFFER_ELEM, hipMemcpyHostToDevice, s0));
    CUDA_CALL(hipMemcpyAsync(
        &d_B[off], &h_B[off], sizeof(float) * NUM_BUFFER_ELEM, hipMemcpyHostToDevice, s0));

    CUDA_CALL(hipEventRecord(ev0, s0));

    CUDA_CALL(hipStreamWaitEvent(s1, ev0));

    size_t numThreads = 256;
    size_t numBlocks = (NUM_BUFFER_ELEM + numThreads - 1) / numThreads;
    fakeVecAdd<<<numBlocks, numThreads, 0, s1>>>(&d_A[off], &d_B[off], &d_C[off], NUM_BUFFER_ELEM);
    CUDA_CALL(hipGetLastError());

    CUDA_CALL(hipEventRecord(ev1, s1));

    CUDA_CALL(hipStreamWaitEvent(s2, ev1));
    CUDA_CALL(hipMemcpyAsync(
        &h_C_buffered[off], &d_C[off], sizeof(float) * NUM_BUFFER_ELEM, hipMemcpyDeviceToHost, s2));
  }

  CUDA_CALL(hipStreamSynchronize(s0));
  CUDA_CALL(hipStreamSynchronize(s1));
  CUDA_CALL(hipStreamSynchronize(s2));
}

int main(int argc, char *argv[])
{
  CUDA_CALL(hipHostMalloc(&h_A, sizeof(float) * NUM_ELEM));
  CUDA_CALL(hipHostMalloc(&h_B, sizeof(float) * NUM_ELEM));
  CUDA_CALL(hipHostMalloc(&h_C_naive, sizeof(float) * NUM_ELEM));
  CUDA_CALL(hipHostMalloc(&h_C_buffered, sizeof(float) * NUM_ELEM));

  CUDA_CALL(hipMalloc(&d_A, sizeof(float) * NUM_ELEM));
  CUDA_CALL(hipMalloc(&d_B, sizeof(float) * NUM_ELEM));
  CUDA_CALL(hipMalloc(&d_C, sizeof(float) * NUM_ELEM));

  CUDA_CALL(hipStreamCreate(&s0));
  CUDA_CALL(hipStreamCreate(&s1));
  CUDA_CALL(hipStreamCreate(&s2));


  CUDA_CALL(hipEventCreate(&ev0));
  CUDA_CALL(hipEventCreate(&ev1));

  for (size_t i = 0; i < NUM_ELEM; ++i) {
    h_A[i] = (float)rand() / RAND_MAX - 0.5;
    h_B[i] = (float)rand() / RAND_MAX - 0.5;
  }

  struct timespec s, e;

  //
  // 1. naive vector addition
  //
  clock_gettime(CLOCK_MONOTONIC, &s);
  fakeVecAdd_naive();
  clock_gettime(CLOCK_MONOTONIC, &e);
  printf("VecAdd_naive: %f ms\n",
      (e.tv_sec - s.tv_sec) * 1000.  + (e.tv_nsec - s.tv_nsec) / 1000000.);

  //
  // 2. double-buffered vector addition
  //
  clock_gettime(CLOCK_MONOTONIC, &s);
  fakeVecAdd_double_buffered();
  clock_gettime(CLOCK_MONOTONIC, &e);
  printf("fakeVecAdd_double_buffered: %f ms\n",
      (e.tv_sec - s.tv_sec) * 1000.  + (e.tv_nsec - s.tv_nsec) / 1000000.);


  printf("Validating ...\n");
  for (size_t i = 0; i < NUM_ELEM; ++i) {
    if (fabs(h_C_naive[i] - h_C_buffered[i]) >= EPS) {
      printf("[%lu] Validation failed: %f %f\n",
          i, h_C_naive[i], h_C_buffered[i]);
      exit(1);
    }
  }
  printf("Validation done\n");
  
  CUDA_CALL(hipHostFree(h_A));
  CUDA_CALL(hipHostFree(h_B));
  CUDA_CALL(hipHostFree(h_C_naive));
  CUDA_CALL(hipHostFree(h_C_buffered));

  CUDA_CALL(hipFree(d_A));
  CUDA_CALL(hipFree(d_B));
  CUDA_CALL(hipFree(d_C));

  CUDA_CALL(hipStreamDestroy(s0));
  CUDA_CALL(hipStreamDestroy(s1));
  CUDA_CALL(hipStreamDestroy(s2));

  CUDA_CALL(hipEventDestroy(ev0));
  CUDA_CALL(hipEventDestroy(ev1));


  return 0;
}
